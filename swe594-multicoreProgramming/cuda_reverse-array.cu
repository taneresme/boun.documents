#include "hip/hip_runtime.h"
#include <iostream>
 
#define N 10

// __global__ qualifier indicates that 
// this function is a kernel function of CUDA.
__global__
void reverse(int *da){
	int tid = blockIdx.x; // which block handling the data 
	if (tid < N){
		int cross = N - 1 - tid;
		int temp = da[tid];
		da[tid] = da[cross];
		da[cross] = temp;
	}
}

int main(int argc, char *argv[])
{	
	std::cout << "Press any button to continue...";
	std::cin.get();
	
	// Define the arrays to be stored on host.
	int A[N], Arev[N];
	// Define the array (pointer) to be stored on device (GPU)
	int *da;
	
	// Fill the array with some values.
	for(int i=0; i<N; i++){
		A[i] = i;//rand() % 100;
		Arev[i] = -1;
	}
	
	// Allocate memory on device for N-item
	hipMalloc((void**)&da, N*sizeof(int));
	// Copy the values on host (A) to device (da)
	// "da" is the memory location to copy A
	hipMemcpy(da, A, N*sizeof(int), hipMemcpyHostToDevice);
	
	// Call kernel with N/2 block (grid).
	reverse<<<N / 2, 1>>>(da);
	
	// Wait for all thread to finish
	hipDeviceSynchronize();
	
	// Copy "da" from device to host (Arev)
	hipMemcpy(Arev, da, N*sizeof(int), hipMemcpyDeviceToHost);
	
	// Print them
	for(int i=0; i<N; i++){
		printf("%d \n", Arev[i]);
	}
	
	// Free the allocated memory on device
	hipFree(da);
	
	return 0;
}
